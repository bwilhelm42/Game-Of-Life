#include "hip/hip_runtime.h"
extern "C" {
    #include "sdl_grid.h"
}

__global__ static void evaluate_cell(int* h_scene, int* updated_scene, int N);
static void render_box(int y, int x, SDL_Renderer* renderer);

extern "C" void    run_simulation_cuda(int* h_scene, SDL_Renderer *renderer, int delay) {
    int *updated_scene, *temp, *scene;
    int x = 0;
    int N = GRID_H * GRID_W;
    SDL_Event event;

    hipMallocManaged(&updated_scene, GRID_H * GRID_W * sizeof(int));
    hipMallocManaged(&scene, GRID_H * GRID_W * sizeof(int));
    hipMemcpy(scene, h_scene, GRID_H * GRID_W * sizeof(int), hipMemcpyHostToDevice);

    while (x++ < NUM_ITERATIONS) {
        while (SDL_PollEvent(&event)) {
            if (event.type == SDL_QUIT || event.key.keysym.sym == SDLK_ESCAPE) {
                SDL_Quit();
                return;
            }
        }
        evaluate_cell<<<GRID_H, GRID_W>>>(scene, updated_scene, N);
        if (delay) {
            scene_update(updated_scene, renderer);
            SDL_Delay(delay);
        }
        temp = updated_scene;
        updated_scene = scene;
        scene = temp;
    }
    hipFree(updated_scene);
    hipFree(scene);
    free(h_scene);
    return;
}

__global__
static void evaluate_cell(int* scene, int* updated_scene, int N) {
    int alive_cells = 0;
    int i = blockIdx.x;
    int j = threadIdx.x;
    
    alive_cells = 
    (i > 0 && j > 0 && scene[(i - 1) * GRID_H + (j - 1)]) +
    (j > 0 && scene[i * GRID_H + (j - 1)]) +
    (i < GRID_H - 1 && j > 0 && scene[(i + 1) * GRID_H + (j - 1)]) +

    (i > 0 && scene[(i - 1) * GRID_H + j]) +
    (i < GRID_H - 1 && scene[(i + 1) * GRID_H + j]) +

    (i > 0 && j < GRID_W - 1 && scene[(i - 1) * GRID_H + (j + 1)]) +
    (j < GRID_W - 1 && scene[i * GRID_H + (j + 1)]) +
    (i < GRID_H - 1 && j < GRID_W - 1 && scene[(i + 1) * GRID_H + (j + 1)]);
    updated_scene[i * GRID_W + j] = (alive_cells == 3 || (alive_cells == 2 && scene[i * GRID_H + j] == 1));
}