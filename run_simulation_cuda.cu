#include "hip/hip_runtime.h"
extern "C" {
    #include "sdl_grid.h"
}

int     calculate_stream_dim(int num_streams, int N);
void    generate_streams(hipStream_t *streams, int stream_count);
__global__ static void evaluate_cell(int* h_scene, int* updated_scene, int N);

extern "C" void    run_simulation_cuda(int* h_scene, SDL_Renderer *renderer, int delay) {
    int *updated_scene, *temp, *scene;
    int x = 0;
    int N = GRID_H * GRID_W;
    SDL_Event event;
    hipError_t error;
    hipStream_t *streams = NULL;
    int index, data_size;
    int num_streams = 1;
    int stream_dim = calculate_stream_dim(num_streams, N);

    generate_streams(streams, num_streams);

    hipMallocManaged(&updated_scene, GRID_H * GRID_W * sizeof(int));
    if ((error = hipGetLastError())) {
        printf("ERROR: %s\n", hipGetErrorString(error));
    }
    hipMallocManaged(&scene, GRID_H * GRID_W * sizeof(int));
    hipMemcpy(scene, h_scene, GRID_H * GRID_W * sizeof(int), hipMemcpyHostToDevice);

    while (x++ < NUM_ITERATIONS) {
        while (SDL_PollEvent(&event)) {
            if (event.type == SDL_QUIT || event.key.keysym.sym == SDLK_ESCAPE) {
                SDL_Quit();
                return;
            }
        }
        for (int i = 0; i < num_streams; i++) {
            index = stream_dim * (i + 1);
            data_size = index > N ? index - N : stream_dim;
            printf("%d\n", (&(scene[i * stream_dim]))[i * stream_dim]);
            printf("Running stream %d\nData size: %d\n", i, data_size);
            evaluate_cell<<<128, 128, 0, streams[i]>>>(scene, updated_scene, data_size);
            printf("made it\n");
        }
        hipDeviceSynchronize();
        hipMemcpy(h_scene, updated_scene, sizeof(int) * GRID_H * GRID_W, hipMemcpyDeviceToHost);
        if (delay) {
            scene_update(h_scene, renderer);
            SDL_Delay(delay);
        }
        temp = scene;
        scene = updated_scene;
        updated_scene = temp;
    }
    hipFree(updated_scene);
    hipFree(scene);
    hipFree(h_scene);
    return;
}

__global__
static void evaluate_cell(int* scene, int* updated_scene, int N) {
    int alive_cells = 0;
    int i = blockIdx.x;
    int j = threadIdx.x;
    
    for (int k = i * blockDim.x + j; k < N; k += blockDim.x * gridDim.x) {

        alive_cells =

        (i > 0 && j > 0 && scene[(i - 1) * GRID_H + (j - 1)]) +
        (j > 0 && scene[i * GRID_H + (j - 1)]) +
        (i < GRID_H - 1 && j > 0 && scene[(i + 1) * GRID_H + (j - 1)]) +

        (i > 0 && scene[(i - 1) * GRID_H + j]) +
        (i < GRID_H - 1 && scene[(i + 1) * GRID_H + j]) +

        (i > 0 && j < GRID_W - 1 && scene[(i - 1) * GRID_H + (j + 1)]) +
        (j < GRID_W - 1 && scene[i * GRID_H + (j + 1)]) +
        (i < GRID_H - 1 && j < GRID_W - 1 && scene[(i + 1) * GRID_H + (j + 1)]);

        updated_scene[i * GRID_W + j] = (alive_cells == 3 || (alive_cells == 2 && scene[i * GRID_H + j] == 1));
    }
}

void    generate_streams(hipStream_t *streams, int stream_count) {
    streams = (hipStream_t*)malloc(sizeof(hipStream_t) * stream_count);
    for (int i = 0; i < stream_count; i++) {
        hipStreamCreate(&(streams[i]));
    }
}

int     calculate_stream_dim(int num_streams, int N) {
    int upper, lower, dim;

    upper = N / num_streams;
    lower = N - (upper * num_streams);
    dim = upper + lower;
    return dim;
}