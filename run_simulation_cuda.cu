#include "hip/hip_runtime.h"
extern "C" {
    #include "sdl_grid.h"
}

__global__ static void evaluate_cell(int* h_scene, int* updated_scene, int N);
static void render_box(int y, int x, SDL_Renderer* renderer);

extern "C" void    run_simulation_cuda(int* h_scene, SDL_Renderer *renderer, int delay) {
    int *updated_scene, *temp, *scene;
    int x = 0;
    int N = GRID_H * GRID_W;

    hipMallocManaged(&updated_scene, GRID_H * GRID_W * sizeof(int));
    hipMallocManaged(&scene, GRID_H * GRID_W * sizeof(int));
    hipMemcpy(scene, h_scene, GRID_H * GRID_W * sizeof(int), hipMemcpyHostToDevice);

    while (x++ < 1000000) {
        evaluate_cell<<<GRID_H, GRID_W>>>(scene, updated_scene, N);
        draw_grid(renderer);
        for (int i = 0; i < N; i++) {
            if (scene[i]) {
                render_box(i / GRID_H, i % GRID_H, renderer);
            }
        }
        SDL_RenderPresent(renderer);
        temp = updated_scene;
        updated_scene = scene;
        scene = temp;
    }
    hipFree(updated_scene);
    hipFree(scene);
    free(h_scene);
    return;
}

static void render_box(int y, int x, SDL_Renderer* renderer) {
	SDL_Color box_color = {.r = 240, .g = 240, .b = 240, .a = 240};
    SDL_Rect rect;
    
    rect.h = CELL_SIZE - 1;
    rect.w = CELL_SIZE - 1;
    rect.x = x * CELL_SIZE;
    rect.y = y * CELL_SIZE;

    SDL_SetRenderDrawColor(renderer, box_color.r, box_color.g, box_color.b, box_color.a);
    SDL_RenderFillRect(renderer, &rect);
}

__global__
static void evaluate_cell(int* scene, int* updated_scene, int N) {
    int alive_cells = 0;
    int i = blockIdx.x;
    int j = threadIdx.x;
    
    alive_cells = 
    (i > 0 && j > 0 && scene[(i - 1) * GRID_H + (j - 1)]) +
    (j > 0 && scene[i * GRID_H + (j - 1)]) +
    (i < GRID_H - 1 && j > 0 && scene[(i + 1) * GRID_H + (j - 1)]) +

    (i > 0 && scene[(i - 1) * GRID_H + j]) +
    (i < GRID_H - 1 && scene[(i + 1) * GRID_H + j]) +

    (i > 0 && j < GRID_W - 1 && scene[(i - 1) * GRID_H + (j + 1)]) +
    (j < GRID_W - 1 && scene[i * GRID_H + (j + 1)]) +
    (i < GRID_H - 1 && j < GRID_W - 1 && scene[(i + 1) * GRID_H + (j + 1)]);
    updated_scene[i * GRID_W + j] = (alive_cells == 3 || (alive_cells == 2 && scene[i * GRID_H + j] == 1));
}