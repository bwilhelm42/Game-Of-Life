#include "hip/hip_runtime.h"
#include "sdl_grid.h"

static int evaluate_cell(int i, int j, int* scene);
static void render_box(int y, int x, SDL_Renderer* renderer);

void    run_simulation_cuda(int* scene, SDL_Renderer *renderer, int delay) {
    int* updated_scene, temp;
    int x = 0;
    int N = GRID_H * GRID_W;

    hipMallocManaged(updated_scene, GRID_H * GRID_W * sizeof(int));

    while (x++ < 100000) {
        evaluate_cell<<<1,1>>>(scene, updated_scene, N);
        temp = updated_scene;
        updated_scene = scene;
        scene = temp;
    }
    free_2d_array(GRID_H, scene);
    free_2d_array(GRID_H, updated_scene);
    return;
}

static void render_box(int y, int x, SDL_Renderer* renderer) {
	SDL_Color box_color = {.r = 240, .g = 240, .b = 240, .a = 240};
    SDL_Rect rect = {
        .h = CELL_SIZE - 1,
        .w = CELL_SIZE - 1,
        .x = x * CELL_SIZE,
        .y = y * CELL_SIZE
    };

    SDL_SetRenderDrawColor(renderer, box_color.r, box_color.g, box_color.b, box_color.a);
    SDL_RenderFillRect(renderer, &rect);
}

__global__
static int evaluate_cell(int* scene, int** updated_scene, int N) {
    int alive_cells = 0;

    alive_cells = 
    (i > 0 && j > 0 && scene[(i - 1) * GRID_H + (j - 1)]) +
    (j > 0 && scene[i * GRID_H + (j - 1)]) +
    (i < GRID_H - 1 && j > 0 && scene[(i + 1) * GRID_H + (j - 1)]) +

    (i > 0 && scene[i - 1][j]) +
    (i < GRID_H - 1 && scene[(i + 1) * GRID_H + j]) +

    (i > 0 && j < GRID_W - 1 && scene[(i - 1) * GRID_H + (j + 1)]) +
    (j < GRID_W - 1 && scene[i * GRID_H + (j + 1)]) +
    (i < GRID_H - 1 && j < GRID_W - 1 && scene[(i + 1) * GRID_H + (j + 1)]);
    updated_scene[i * GRID_W + j] = (alive_cells == 3 || (alive_cells == 2 && scene[i * GRID_H + j] == 1));
}