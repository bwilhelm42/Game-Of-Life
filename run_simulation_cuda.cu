#include "hip/hip_runtime.h"
extern "C" {
    #include "sdl_grid.h"
}

__global__ static void evaluate_cell(int* h_scene, int* updated_scene, int N);

extern "C" void    run_simulation_cuda(int* h_scene, SDL_Renderer *renderer, int delay) {
    int *updated_scene, *temp, *scene;
    int x = 0;
    int N = GRID_H * GRID_W;
    SDL_Event event;
    hipError_t error;

    hipMallocManaged(&updated_scene, GRID_H * GRID_W * sizeof(int));
    if ((error = hipGetLastError())) {
        printf("ERROR: %s\n", hipGetErrorString(error));
    }
    hipMallocManaged(&scene, GRID_H * GRID_W * sizeof(int));
    hipMemcpy(scene, h_scene, GRID_H * GRID_W * sizeof(int), hipMemcpyHostToDevice);

    while (x++ < NUM_ITERATIONS) {
        while (SDL_PollEvent(&event)) {
            if (event.type == SDL_QUIT || event.key.keysym.sym == SDLK_ESCAPE) {
                SDL_Quit();
                return;
            }
        }
        evaluate_cell<<<128, 128>>>(scene, updated_scene, N);
        hipMemcpy(h_scene, updated_scene, sizeof(int) * GRID_H * GRID_W, hipMemcpyDeviceToHost);
        if (delay) {
            scene_update(h_scene, renderer);
            SDL_Delay(delay);
        }
        temp = scene;
        scene = updated_scene;
        updated_scene = temp;
    }
    hipFree(updated_scene);
    hipFree(scene);
    hipFree(h_scene);
    return;
}

__global__
static void evaluate_cell(int* scene, int* updated_scene, int N) {
    int alive_cells = 0;
    int i = blockIdx.x;
    int j = threadIdx.x;
    
    for (int k = i * blockDim.x + j; k < N; k += blockDim.x * gridDim.x) {

        alive_cells =

        (i > 0 && j > 0 && scene[(i - 1) * GRID_H + (j - 1)]) +
        (j > 0 && scene[i * GRID_H + (j - 1)]) +
        (i < GRID_H - 1 && j > 0 && scene[(i + 1) * GRID_H + (j - 1)]) +

        (i > 0 && scene[(i - 1) * GRID_H + j]) +
        (i < GRID_H - 1 && scene[(i + 1) * GRID_H + j]) +

        (i > 0 && j < GRID_W - 1 && scene[(i - 1) * GRID_H + (j + 1)]) +
        (j < GRID_W - 1 && scene[i * GRID_H + (j + 1)]) +
        (i < GRID_H - 1 && j < GRID_W - 1 && scene[(i + 1) * GRID_H + (j + 1)]);

        updated_scene[i * GRID_W + j] = (alive_cells == 3 || (alive_cells == 2 && scene[i * GRID_H + j] == 1));
    }
}