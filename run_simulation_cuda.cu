#include "hip/hip_runtime.h"
extern "C" {
    #include "sdl_grid.h"
}

static void my_swap(int **a, int **b);
__global__ static void evaluate_cell(int* h_scene, int* updated_scene, int N);

extern "C" void    run_simulation_cuda(int* h_scene, SDL_Renderer *renderer, int delay) {
    int *updated_scene, *scene;
    int x = 0;
    int N = GRID_H * GRID_W;
    SDL_Event event;
    hipError_t error;

    hipMalloc(&updated_scene, GRID_H * GRID_W * sizeof(int));
    hipMalloc(&scene, GRID_H * GRID_W * sizeof(int));
    hipMemcpy(scene, h_scene, GRID_H * GRID_W * sizeof(int), hipMemcpyHostToDevice);

    while (x++ < NUM_ITERATIONS) {
        while (SDL_PollEvent(&event)) {
            if (event.type == SDL_QUIT || event.key.keysym.sym == SDLK_ESCAPE) {
                SDL_Quit();
                return;
            }
        }
        evaluate_cell<<<GRID_W, GRID_H>>>(scene, updated_scene, N);
        hipMemcpy(h_scene, updated_scene, sizeof(int) * GRID_H * GRID_W, hipMemcpyDeviceToHost);
        if ((error = hipGetLastError())) {
            printf("ERROR: %s\n", hipGetErrorString(error));
        }
        if (delay) {
            scene_update(h_scene, renderer);
            SDL_Delay(delay);
        }
        my_swap(&scene, &updated_scene);
    }
    hipFree(updated_scene);
    hipFree(scene);
    return;
}

__global__
static void evaluate_cell(int* scene, int* updated_scene, int N) {
    int alive_cells = 0;
    int i = blockIdx.x;
    int j = threadIdx.x;
    
    for (int k = i * blockDim.x + j; k < N; k += blockDim.x * gridDim.x) {

        alive_cells =

        (i > 0 && j > 0 && scene[(i - 1) * GRID_H + (j - 1)]) +
        (j > 0 && scene[i * GRID_H + (j - 1)]) +
        (i < GRID_H - 1 && j > 0 && scene[(i + 1) * GRID_H + (j - 1)]) +

        (i > 0 && scene[(i - 1) * GRID_H + j]) +
        (i < GRID_H - 1 && scene[(i + 1) * GRID_H + j]) +

        (i > 0 && j < GRID_W - 1 && scene[(i - 1) * GRID_H + (j + 1)]) +
        (j < GRID_W - 1 && scene[i * GRID_H + (j + 1)]) +
        (i < GRID_H - 1 && j < GRID_W - 1 && scene[(i + 1) * GRID_H + (j + 1)]);

        updated_scene[i * GRID_W + j] = (alive_cells == 3 || (alive_cells == 2 && scene[i * GRID_H + j] == 1));
    }
}

static void my_swap(int **a, int **b) {
    int *c;

    c = *a;
    *a = *b;
    *b = c;
}