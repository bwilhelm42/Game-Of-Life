#include "hip/hip_runtime.h"
extern "C" {
    #include "sdl_grid.h"
}

int main() {
	SDL_Window *window;
	SDL_Renderer *renderer;
	int*	scene;
	struct runtime_info info;

	if (get_run_mode(&info) == -1) {
		return -1;
	}
	if (SDL_Init(SDL_INIT_VIDEO) == -1){
		printf("Couldn't initialize SDL2: %s", SDL_GetError());
        return -1;
	}

	hipHostMalloc(&scene, sizeof(int) * GRID_W * GRID_H);
	bzero(scene, sizeof(int) * GRID_W * GRID_H);
	if (info.delay > 0) {
		if (SDL_CreateWindowAndRenderer(WINDOW_W, WINDOW_H, 0, &window, &renderer) == -1) {
			printf("Could not initialize window: %s", SDL_GetError());
			return -1;
		}
		init_world(scene, renderer);
	}

	clock_t t;
	t = clock();

	switch (info.device) {
		case CPU:
			run_simulation(scene, renderer, info.delay);
			break;
		case GPU:
			run_simulation_cuda(scene, renderer, info.delay);
			break;
		default:
			break;
	}
    hipFree(scene);

	t = clock() - t;
	double time = (double)t / CLOCKS_PER_SEC;
	printf("Time spent: %f\n", time);

	return 0;
}