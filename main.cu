#include "hip/hip_runtime.h"
extern "C" {
    #include "sdl_grid.h"
}

int main() {
	SDL_Window *window;
	SDL_Renderer *renderer;
	int*	scene;
	struct runtime_info info;
	hipError_t error;

	if (get_run_mode(&info) == -1) {
		return -1;
	}
	if (SDL_Init(SDL_INIT_VIDEO) == -1){
		printf("Couldn't initialize SDL2: %s", SDL_GetError());
        return -1;
	}

	hipHostMalloc(&scene, sizeof(int) * GRID_W * GRID_H);
	if ((error = hipGetLastError())) {
		printf("ERROR1: %s\n", hipGetErrorString(error));
	}
	if (info.delay > 0) {
		if (SDL_CreateWindowAndRenderer(WINDOW_W, WINDOW_H, 0, &window, &renderer) == -1) {
			printf("Could not initialize window: %s", SDL_GetError());
			return -1;
		}
		init_world(scene, renderer);
	}

	clock_t t;
	t = clock();

	switch (info.device) {
		case CPU:
			run_simulation(scene, renderer, info.delay);
			hipFree(scene); //Can't free in function since it is in C not CU
			break;
		case GPU:
			run_simulation_cuda(scene, renderer, info.delay);
			break;
		default:
			break;
	}
	t = clock() - t;
	double time = (double)t / CLOCKS_PER_SEC;
	printf("Time spent: %f\n", time);

	return 0;
}